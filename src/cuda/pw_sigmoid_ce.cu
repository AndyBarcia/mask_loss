#include "hip/hip_runtime.h"
#include <ATen/native/cuda/KernelUtils.cuh>
#include <c10/util/Half.h>
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <cmath>

#include "utils.h"

// Process regions of 16x16, perfect for logits of shape
// 64x64 and ground truth of shape 1024x1024.
const int HIGH_RES_BLOCK = 16;
const int COUNTER_THREADS_PER_BLOCK = HIGH_RES_BLOCK*HIGH_RES_BLOCK;
const int REDUCTION_THREADS_PER_BLOCK = 256;

template <int H, int W, int H_t, int W_t>
__global__ void __launch_bounds__(COUNTER_THREADS_PER_BLOCK) count_labels_per_block_kernel(
    const int64_t* __restrict__ targets,
    int32_t* __restrict__ counts, // out: shape (B, GT_total, H, W)
    const int B,
    const int GT_total
) {
    extern __shared__ int32_t sh_mem_counts[];
    int* sh_counts = sh_mem_counts;

    int j = blockIdx.x;  // low-res x
    int i = blockIdx.y;  // low-res y
    int b = blockIdx.z;  // batch

    int tid = threadIdx.x;
    const int s = H_t / H;
    const int s2 = s * s;

    // Initialize shared memory counts for this block
    for (int idx = tid; idx < GT_total; idx += COUNTER_THREADS_PER_BLOCK) {
        sh_counts[idx] = 0;
    }
    __syncthreads();

    // Base corner of the corresponding high-resolution block
    int base_y = i * s;
    int base_x = j * s;

    // Parallel count of pixels per GT label within the block
    for (int idx = tid; idx < s2; idx += COUNTER_THREADS_PER_BLOCK) {
        int dy = idx / s;
        int dx = idx % s;
        int yy = base_y + dy;
        int xx = base_x + dx;
        if (yy < H_t && xx < W_t) {
            int64_t lab = targets[(b * H_t + yy) * W_t + xx];
            if (lab >= 0 && lab < GT_total) {
                atomicAdd(&sh_counts[(int)lab], 1);
            }
        }
    }
    __syncthreads();

    // Write the counts from shared memory to the global counts tensor
    for (int gt = tid; gt < GT_total; gt += COUNTER_THREADS_PER_BLOCK) {
        counts[((b * GT_total + gt) * H + i) * W + j] = sh_counts[gt];
    }
}

template <int C, int H, int W, int H_t>
__global__ void __launch_bounds__(REDUCTION_THREADS_PER_BLOCK) reduce_loss_kernel(
    const float* __restrict__ logits,           // shape (L, B, C, H, W)
    const int32_t* __restrict__ counts,         // shape (B, GT_total, H, W)
    float* __restrict__ out,                   // shape (L, B, C, GT_out)
    const int32_t* __restrict__ total_counts,   // shape (B, GT_total)
    const int32_t* __restrict__ gt_map,         // length GT_out: maps output index -> actual GT label
    const int GT_total,
    const int GT_out,
    const int B,
    const int L
) {
    extern __shared__ float s_block_loss[];

    const int out_gt_idx = blockIdx.x; // compacted output index (0..GT_out-1)
    const int ci = blockIdx.y;         // Logit class index
    const int flat_b_l = blockIdx.z;   // flat index combining layer and batch: 0 .. (B*L - 1)

    // Recover layer and batch indices
    const int b = flat_b_l % B;
    const int l = flat_b_l / B;

    // Map to the actual ground-truth label index
    const int gt_actual = gt_map[out_gt_idx];

    // If the total count for this ground truth label is 0, it's a zero-area mask.
    // Set the loss to infinity and return. Only one thread writes the output
    if (total_counts[b * GT_total + gt_actual] == 0) {
        if (threadIdx.x == 0) {
            out[((l*B + b)*C + ci) * GT_out + out_gt_idx] = INFINITY;
        }
        return;
    }

    const int tid = threadIdx.x;
    const int s = H_t / H;
    const float N2 = static_cast<float>(s * s);

    float thread_loss_sum = 0.0;

    // Each thread computes a partial sum of the loss over the HxW logit plane
    for (int idx = tid; idx < H * W; idx += REDUCTION_THREADS_PER_BLOCK) {
        int i = idx / W;
        int j = idx % W;

        float L = logits[(((l * B + b) * C + ci) * H + i) * W + j];
        int32_t n = counts[((b * GT_total + gt_actual) * H + i) * W + j];

        float maxL = L > 0.0f ? L : 0.0f;
        float absL = fabsf(L);
        float logexp = my_log1pf(__expf(-absL));

        // This follows: sum_{pixels in HR patch} [ max(L,0) - L*y + log(1+exp(-|L|)) ]
        // where y is the per-pixel binary GT. Here n is the count of y==1 in the HR patch,
        // and N2 is number of HR pixels in the patch.
        thread_loss_sum += (N2 * (maxL + logexp) - L * n);
    }

    // Warp-level-reduction: Sum thread-level losses within each warp.
    #pragma unroll
    for (int offset = 16; offset > 0; offset >>= 1) { // 16 is half the warp size
        thread_loss_sum += __shfl_down_sync(0xffffffff, thread_loss_sum, offset);
    }

    // The first thread (lane 0) of each warp writes the warp's sum to shared memory.
    const int lane_id = tid % 32;
    const int warp_id = tid / 32;
    if (lane_id == 0) {
        s_block_loss[warp_id] = thread_loss_sum;
    }

    // Store each thread's accumulated loss into shared memory
    //s_block_loss[tid] = thread_loss_sum;
    __syncthreads();

    // Perform the block-level reduction
    const int num_warps = REDUCTION_THREADS_PER_BLOCK / 32;
    for (int s_reduce = num_warps / 2; s_reduce > 0; s_reduce >>= 1) {
        if (tid < s_reduce) {
            s_block_loss[tid] += s_block_loss[tid + s_reduce];
        }
        __syncthreads();
    }

    // The first thread writes the final reduced result for the block, no atomic needed
    if (tid == 0) {
        out[((l * B + b) * C + ci) * GT_out + out_gt_idx] = s_block_loss[0];
    }
}

torch::Tensor pairwise_sigmoid_cross_entropy_forward(
    const torch::Tensor& logits,
    const torch::Tensor& targets,
    const int64_t background_index = -1
) {
    CHECK_INPUT(logits);
    CHECK_INPUT(targets);

    const int L = logits.size(0);
    const int B = logits.size(1);
    const int C = logits.size(2);
    const int H = logits.size(3);
    const int W = logits.size(4);
    const int H_t = targets.size(1);
    const int W_t = targets.size(2);

    // Automatically compute GT_total from the max value in targets
    torch::Tensor targets_max_tensor = targets.max();
    const int64_t GT_total_64 = targets_max_tensor.item<int64_t>() + 1;
    const int GT_total = static_cast<int>(GT_total_64);

    // Intermediate tensor to store counts for every possible GT label (including background)
    auto counts = torch::zeros({B, GT_total, H, W}, logits.options().dtype(torch::kInt32));

    // Launch count kernel (counts for ALL GT labels)
    {
        dim3 grid(W, H, B);
        const size_t shared_mem_size = GT_total * sizeof(int32_t);

        auto static_launcher = [&](auto H_val, auto W_val, auto H_t_val, auto W_t_val) {
            count_labels_per_block_kernel<
                decltype(H_val)::value, decltype(W_val)::value,
                decltype(H_t_val)::value, decltype(W_t_val)::value>
                <<<grid, COUNTER_THREADS_PER_BLOCK, shared_mem_size>>>(
                    targets.data_ptr<int64_t>(),
                    counts.data_ptr<int32_t>(),
                    B, GT_total
                );
        };
        const auto supported_dims = std::make_tuple(
            std::make_tuple(std::integral_constant<int, 64>{}),  // H
            std::make_tuple(std::integral_constant<int, 64>{}),  // W
            std::make_tuple(std::integral_constant<int, 512>{}, std::integral_constant<int, 1024>{}), // H_t
            std::make_tuple(std::integral_constant<int, 512>{}, std::integral_constant<int, 1024>{})  // W_t
        );
        const auto runtime_dims = std::make_tuple(H, W, H_t, W_t);
        dispatch_kernel(static_launcher, runtime_dims, supported_dims);
    }
    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA error after count kernel: ", hipGetErrorString(err));

    // Calculate the total number of pixels for each ground truth label (mask area)
    // This is used to mask 0-area masks with a loss of infinity.
    auto total_counts = counts.sum({2, 3}).to(torch::kInt32).contiguous(); // shape (B, GT_total)

    // Build gt_map: list of actual GT labels to evaluate (exclude background_index if requested and valid)
    std::vector<int32_t> host_gt_map;
    host_gt_map.reserve(GT_total);
    for (int i = 0; i < GT_total; ++i) {
        if (background_index >= 0 && i == static_cast<int>(background_index)) {
            continue;
        }
        host_gt_map.push_back(i);
    }
    const int GT_out = static_cast<int>(host_gt_map.size());
    // If no classes left to evaluate (edge case), return an empty tensor of shape (B,C,0)
    if (GT_out == 0) {
        return torch::zeros({L, B, C, 0}, logits.options().dtype(torch::kFloat32));
    }

    // Copy gt_map to device
    auto gt_map = torch::from_blob(host_gt_map.data(), {GT_out}, torch::kInt32).clone().to(logits.device());

    // Launch reduction kernel only for the compacted GT_out entries (mapping via gt_map)
    auto out_accum = torch::zeros({L, B, C, GT_out}, logits.options().dtype(torch::kFloat32));
    {
        dim3 grid(GT_out, C, B*L);
        const size_t shared_mem_size = (REDUCTION_THREADS_PER_BLOCK / 32) * sizeof(float);

        auto static_launcher = [&](auto C_val, auto H_val, auto W_val, auto H_t_val) {
            reduce_loss_kernel<
                decltype(C_val)::value, decltype(H_val)::value,
                decltype(W_val)::value, decltype(H_t_val)::value>
                <<<grid, REDUCTION_THREADS_PER_BLOCK, shared_mem_size>>>(
                    logits.data_ptr<float>(),
                    counts.data_ptr<int32_t>(),
                    out_accum.data_ptr<float>(),
                    total_counts.data_ptr<int32_t>(),
                    gt_map.data_ptr<int32_t>(),
                    GT_total,
                    GT_out,
                    B, L
                );
        };

        const auto supported_dims = std::make_tuple(
            std::make_tuple(std::integral_constant<int, 256>{}), // C
            std::make_tuple(std::integral_constant<int, 64>{}),  // H
            std::make_tuple(std::integral_constant<int, 64>{}),  // W
            std::make_tuple(std::integral_constant<int, 512>{}, std::integral_constant<int, 1024>{}) // H_t
        );
        // W_t is not needed by reduce_loss_kernel, so we only pass needed dims
        const auto runtime_dims = std::make_tuple(C, H, W, H_t);
        dispatch_kernel(static_launcher, runtime_dims, supported_dims);
    }

    err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA error after reduce kernel: ", hipGetErrorString(err));

    // Normalize by total high-res pixels per block (H_t * W_t)
    auto out_final = out_accum.to(logits.options().dtype(torch::kFloat32)) / static_cast<float>(H_t * W_t);

    return out_final;
}