#include "hip/hip_runtime.h"
#include <ATen/native/cuda/KernelUtils.cuh>
#include <c10/util/Half.h>
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <cmath>

#include "utils.h"

const int THREADS_PER_BLOCK = 256;

template <int C, int H, int W, int H_t, int W_t>
__global__ void __launch_bounds__(THREADS_PER_BLOCK, 2) sigmoid_cross_entropy_forward_kernel(
    const float* __restrict__ logits,
    const int64_t* __restrict__ targets,
    double* __restrict__ total_loss_sum,
    const int B
) {
    // Each CUDA block processes one (b, i, j) low-res block
    // Grid: dim.x = W (low-res width), dim.y = H (low-res height), dim.z = B (batch)
    // threadIdx.x loops over s*s pixels
    extern __shared__ int sh_counts[]; // Shared memory for per-class counts, size C

    int j = blockIdx.x;  // low-res x (0..W-1)
    int i = blockIdx.y;  // low-res y (0..H-1)
    int b = blockIdx.z;  // batch index

    int tid = threadIdx.x;
    const int s = H_t / H; // Stride
    const int s2 = s * s;

    // Initialize shared counts to zero
    for (int ci = tid; ci < C; ci += THREADS_PER_BLOCK) {
        sh_counts[ci] = 0;
    }
    __syncthreads();

    // Each thread block covers an s x s region of the high-resolution target tensor.
    // Top-left corner of the high-res block:
    int base_y = i * s;
    int base_x = j * s;

    // Each thread loops over several pixels if necessary
    for (int idx = tid; idx < s2; idx += THREADS_PER_BLOCK) {
        int dy = idx / s;
        int dx = idx % s;
        int yy = base_y + dy;
        int xx = base_x + dx;
        if (yy < H_t && xx < W_t) {
            // targets layout: (B, H_t, W_t)
            int64_t lab = targets[(b * H_t + yy) * W_t + xx];
            if (lab >= 0 && lab < C) {
                // Atomically accumulate counts in shared memory
                atomicAdd(&sh_counts[(int)lab], 1);
            }
        }
    }
    __syncthreads();

    // Each thread computes the loss for a subset of classes
    for (int ci = tid; ci < C; ci += THREADS_PER_BLOCK) {
        // logits layout: (B, C, H, W)
        float L = logits[((b * C + ci) * H + i) * W + j];
        float n = (float) sh_counts[ci];
        float N2 = (float)s2;

        // Stable BCE-with-logits sum over the block:
        // loss_block = N2*max(L,0) - L*n + N2*log1p(exp(-|L|))
        float maxL = L > 0.0f ? L : 0.0f;
        float absL = fabsf(L);
        float logexp = log1pf(expf(-absL));
        double loss_block = static_cast<double>(N2 * maxL - L * n + N2 * logexp);

        // Atomically add the block's loss to the total sum
        atomicAdd(total_loss_sum, loss_block);
    }
}

template <int C, int H, int W, int H_t, int W_t>
__global__ void __launch_bounds__(THREADS_PER_BLOCK, 2) sigmoid_cross_entropy_backward_kernel(
    const float* __restrict__ logits,
    const int64_t* __restrict__ targets,
    const float grad_out_scalar,
    float* __restrict__ grad_logits,
    const int B
) {
    // Grid: dim.x = W (low-res x), dim.y = H (low-res y), dim.z = B
    int j = blockIdx.x;
    int i = blockIdx.y;
    int b = blockIdx.z;

    // Parallelize across classes in threads
    int tid = threadIdx.x;

    const int s = H_t / H; // Stride
    const float N2 = (float)(s * s);

    // To calculate the gradient, we need to re-compute the counts for each block.
    // This is a trade-off to avoid storing the counts tensor from the forward pass.
    // Shared memory is used for efficient recounting.
    extern __shared__ int sh_counts[]; // Shared memory for per-class counts, size C

    // Initialize shared counts to zero
    for (int ci = tid; ci < C; ci += THREADS_PER_BLOCK) {
        sh_counts[ci] = 0;
    }
    __syncthreads();

    // Re-compute counts for the current block
    int base_y = i * s;
    int base_x = j * s;
    for (int idx = tid; idx < N2; idx += THREADS_PER_BLOCK) {
        int dy = idx / s;
        int dx = idx % s;
        int yy = base_y + dy;
        int xx = base_x + dx;
        if (yy < H_t && xx < W_t) {
            int64_t lab = targets[(b * H_t + yy) * W_t + xx];
            if (lab >= 0 && lab < C) {
                atomicAdd(&sh_counts[(int)lab], 1);
            }
        }
    }
    __syncthreads();
    
    // Base index for the current block
    int idx_base = ((b * C) * H + i) * W + j;
    float scale = grad_out_scalar / (B * C * H * W);

    // Each thread computes the gradient for a subset of classes
    for (int ci = tid; ci < C; ci += THREADS_PER_BLOCK) {
        float L = logits[idx_base + ci * H * W];
        int32_t n = sh_counts[ci];

        // sigma = 1 / (1 + exp(-L))
        float sigma = 1.0f / (1.0f + expf(-L));
        // derivative: dLoss/dL = N2 * sigma - n
        float g = N2 * sigma - (float)n;
        
        // Apply scaling
        grad_logits[idx_base + ci * H * W] = g * scale;
    }
}

torch::Tensor sigmoid_cross_entropy_forward(
    const torch::Tensor& logits, 
    const torch::Tensor& targets
) {
    CHECK_INPUT(logits);
    CHECK_INPUT(targets);

    const int B = logits.size(0);
    const int C = logits.size(1);
    const int H = logits.size(2);
    const int W = logits.size(3);
    const int H_t = targets.size(1);
    const int W_t = targets.size(2);
    
    const int total_elements = B * C * H_t * W_t;
    if (total_elements == 0) return torch::tensor(0.0, logits.options());

    auto total_loss_sum_tensor = torch::zeros({1}, logits.options().dtype(torch::kFloat64));
    
    // Set grid dimensions based on the low-resolution output
    dim3 grid(W, H, B);
    
    // Shared memory size is C * sizeof(int32) for the counts
    const size_t shared_mem_size = C * sizeof(int32_t);

    auto static_launcher = [&](auto... Dims) {
        sigmoid_cross_entropy_forward_kernel<decltype(Dims)::value...><<<grid, THREADS_PER_BLOCK, shared_mem_size>>>(
            logits.data_ptr<float>(), targets.data_ptr<int64_t>(), total_loss_sum_tensor.data_ptr<double>(), B);
    };

    const auto supported_dims = std::make_tuple(
        std::make_tuple(std::integral_constant<int, 256>{}), // C
        std::make_tuple(std::integral_constant<int, 64>{}),  // H
        std::make_tuple(std::integral_constant<int, 64>{}),  // W
        std::make_tuple(std::integral_constant<int, 512>{}), // H_t
        std::make_tuple(std::integral_constant<int, 512>{})  // W_t
    );
    const auto runtime_dims = std::make_tuple(C, H, W, H_t, W_t);

    dispatch_kernel(static_launcher, runtime_dims, supported_dims);

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA error after forward kernel: ", hipGetErrorString(err));
    
    return (total_loss_sum_tensor.to(torch::kFloat32) / total_elements).squeeze();
}

torch::Tensor sigmoid_cross_entropy_backward(
    const torch::Tensor& grad_out, 
    const torch::Tensor& logits, 
    const torch::Tensor& targets
) {
    
    CHECK_INPUT(grad_out);
    CHECK_INPUT(logits);
    CHECK_INPUT(targets);

    const int B = logits.size(0);
    const int C = logits.size(1);
    const int H = logits.size(2);
    const int W = logits.size(3);
    const int H_t = targets.size(1);
    const int W_t = targets.size(2);

    auto grad_logits = torch::empty_like(logits);
    const int total_elements = B * C * H * W;
    if (total_elements == 0) return grad_logits;

    const float grad_out_scalar = grad_out.item<float>();
    
    // Set grid dimensions based on the low-resolution output
    dim3 grid(W, H, B);
    const size_t shared_mem_size = C * sizeof(int32_t);

    auto static_launcher = [&](auto... Dims) {
        sigmoid_cross_entropy_backward_kernel<decltype(Dims)::value...><<<grid, THREADS_PER_BLOCK, shared_mem_size>>>(
            logits.data_ptr<float>(), targets.data_ptr<int64_t>(), grad_out_scalar, grad_logits.data_ptr<float>(), B);
    };
    
    const auto supported_dims = std::make_tuple(
        std::make_tuple(std::integral_constant<int, 256>{}), // C
        std::make_tuple(std::integral_constant<int, 64>{}),  // H
        std::make_tuple(std::integral_constant<int, 64>{}),  // W
        std::make_tuple(std::integral_constant<int, 512>{}), // H_t
        std::make_tuple(std::integral_constant<int, 512>{})  // W_t
    );
    const auto runtime_dims = std::make_tuple(C, H, W, H_t, W_t);

    dispatch_kernel(static_launcher, runtime_dims, supported_dims);

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA error after backward kernel: ", hipGetErrorString(err));

    return grad_logits;
}