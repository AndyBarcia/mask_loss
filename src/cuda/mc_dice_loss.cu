#include "hip/hip_runtime.h"
#include <ATen/native/cuda/KernelUtils.cuh>
#include <c10/util/Half.h>
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <cmath>

#include "utils.h"

// Process regions of 16x16, perfect for logits of shape
// 64x64 and ground truth of shape 1024x1024.
const int THREADS_PER_BLOCK = 16 * 16;

template <int C, int H, int W, int H_t, int W_t>
__global__ void __launch_bounds__(THREADS_PER_BLOCK, 2) dice_loss_forward_kernel(
    const float* __restrict__ logits,
    const uint8_t* __restrict__ targets,
    const int64_t* __restrict__ class_mapping,
    float* __restrict__ total_intersection_sum,
    float* __restrict__ total_p_sum,
    float* __restrict__ total_t_sum,
    const int B
) {
    // Each CUDA block processes one (b, i, j) low-res block
    // Grid: dim.x = W (low-res width), dim.y = H (low-res height), dim.z = B (batch)
    extern __shared__ char sh_mem[];
    int* sh_counts = reinterpret_cast<int*>(sh_mem);
    int64_t* sh_class_mapping = reinterpret_cast<int64_t*>(sh_mem + C * sizeof(int));

    int j = blockIdx.x; // low-res x (0..W-1)
    int i = blockIdx.y; // low-res y (0..H-1)
    int b = blockIdx.z; // batch index

    int tid = threadIdx.x;
    const int s = H_t / H; // Stride
    const int s2 = s * s;

    // Initialize shared counts to zero
    for (int ci = tid; ci < C; ci += THREADS_PER_BLOCK) {
        sh_counts[ci] = 0;
    }
    // Collaboratively load class_mapping into shared memory.
    // Since THREADS_PER_BLOCK is 256, each thread loads exactly one value.
    if (tid < 256) {
        sh_class_mapping[tid] = class_mapping[tid];
    }
    __syncthreads();

    // Each thread block covers an s x s region of the high-resolution target tensor.
    // Top-left corner of the high-res block:
    int base_y = i * s;
    int base_x = j * s;

    // Each thread loops over several pixels if necessary to compute counts
    for (int idx = tid; idx < s2; idx += THREADS_PER_BLOCK) {
        int dy = idx / s;
        int dx = idx % s;
        int yy = base_y + dy;
        int xx = base_x + dx;
        if (yy < H_t && xx < W_t) {
            // targets layout: (B, H_t, W_t)
            uint8_t raw_lab = targets[(b * H_t + yy) * W_t + xx];
            int64_t lab = sh_class_mapping[raw_lab]; // Read from shared memory
            if (lab >= 0 && lab < C) {
                // Atomically accumulate counts in shared memory
                atomicAdd(&sh_counts[(int)lab], 1);
            }
        }
    }
    __syncthreads();

    // Each thread computes the intersection and sums for a subset of classes
    for (int ci = tid; ci < C; ci += THREADS_PER_BLOCK) {
        // logits layout: (B, C, H, W)
        float L = logits[((b * C + ci) * H + i) * W + j];
        float p = 1.0f / (1.0f + expf(-L));
        float n_k = (float)sh_counts[ci];
        float N2 = (float)s2;

        float intersection = p * n_k;
        float p_sum = N2 * p;
        float t_sum = n_k;

        atomicAdd(&total_intersection_sum[b * C + ci], intersection);
        atomicAdd(&total_p_sum[b * C + ci], p_sum);
        atomicAdd(&total_t_sum[b * C + ci], t_sum);
    }
}

template <int C, int H, int W, int H_t, int W_t>
__global__ void __launch_bounds__(THREADS_PER_BLOCK, 2) dice_loss_backward_kernel(
    const float* __restrict__ logits,
    const uint8_t* __restrict__ targets,
    const int64_t* __restrict__ class_mapping,
    const float* __restrict__ total_intersection_sum,
    const float* __restrict__ total_p_sum,
    const float* __restrict__ total_t_sum,
    const float grad_out_scalar,
    float* __restrict__ grad_logits,
    const int B,
    const float smooth
) {
    // Grid: dim.x = W (low-res x), dim.y = H (low-res y), dim.z = B
    int j = blockIdx.x;
    int i = blockIdx.y;
    int b = blockIdx.z;

    int tid = threadIdx.x;

    const int s = H_t / H;
    const float N2 = (float)(s * s);

    extern __shared__ char sh_mem[];
    int* sh_counts = reinterpret_cast<int*>(sh_mem);
    int64_t* sh_class_mapping = reinterpret_cast<int64_t*>(sh_mem + C * sizeof(int));

    // Initialize shared counts to zero
    for (int ci = tid; ci < C; ci += THREADS_PER_BLOCK) {
        sh_counts[ci] = 0;
    }
    // Collaboratively load class_mapping into shared memory.
    // Since THREADS_PER_BLOCK is 256, each thread loads exactly one value.
    if (tid < 256) {
        sh_class_mapping[tid] = class_mapping[tid];
    }
    __syncthreads();

    // Re-compute counts for the current block.
    int base_y = i * s;
    int base_x = j * s;
    for (int idx = tid; idx < N2; idx += THREADS_PER_BLOCK) {
        int dy = idx / s;
        int dx = idx % s;
        int yy = base_y + dy;
        int xx = base_x + dx;
        if (yy < H_t && xx < W_t) {
            // targets layout: (B, H_t, W_t)
            uint8_t raw_lab = targets[(b * H_t + yy) * W_t + xx];
            int64_t lab = sh_class_mapping[raw_lab]; // Read from shared memory
            if (lab >= 0 && lab < C) {
                atomicAdd(&sh_counts[(int)lab], 1);
            }
        }
    }
    __syncthreads();

    // Each thread computes the gradient for a subset of classes
    float scale = -grad_out_scalar / (B * C);
    for (int ci = tid; ci < C; ci += THREADS_PER_BLOCK) {
        float L = logits[((b * C + ci) * H + i) * W + j];
        float p = 1.0f / (1.0f + expf(-L));
        float n_k = (float)sh_counts[ci];

        float I = total_intersection_sum[b * C + ci];
        float P = total_p_sum[b * C + ci];
        float T = total_t_sum[b * C + ci];

        float denominator = P + T + smooth;
        float term1_numerator = 2.0 * n_k * (P + T + smooth);
        float term2_numerator = 2.0 * N2 * (2.0 * I + smooth);

        float d_dice_dp = (term1_numerator - term2_numerator) / (denominator * denominator);
        float dp_dL = p * (1.0 - p);

        grad_logits[((b * C + ci) * H + i) * W + j] = scale * d_dice_dp * dp_dL;
    }
}

std::vector<torch::Tensor> mc_dice_loss_forward(
    const torch::Tensor& logits,
    const torch::Tensor& targets,
    const torch::Tensor& class_mapping,
    const float smooth,
    const int num_masks
) {
    CHECK_INPUT(logits);
    CHECK_INPUT(targets);

    const int B = logits.size(0);
    const int C = logits.size(1);
    const int H = logits.size(2);
    const int W = logits.size(3);
    const int H_t = targets.size(1);
    const int W_t = targets.size(2);

    if (logits.numel() == 0) {
        return {
            torch::tensor(0.0, logits.options()),
            torch::zeros({B, C}, logits.options()),
            torch::zeros({B, C}, logits.options()),
            torch::zeros({B, C}, logits.options())
        };
    }

    auto total_intersection_sum = torch::zeros({B, C}, logits.options());
    auto total_p_sum = torch::zeros({B, C}, logits.options());
    auto total_t_sum = torch::zeros({B, C}, logits.options());

    dim3 grid(W, H, B);
    const size_t shared_mem_size = 2 * C * sizeof(int32_t);

    auto static_launcher = [&](auto... Dims) {
        dice_loss_forward_kernel<decltype(Dims)::value...><<<grid, THREADS_PER_BLOCK, shared_mem_size>>>(
            logits.data_ptr<float>(),
            targets.data_ptr<uint8_t>(),
            class_mapping.data_ptr<int64_t>(),
            total_intersection_sum.data_ptr<float>(),
            total_p_sum.data_ptr<float>(),
            total_t_sum.data_ptr<float>(),
            B
        );
    };

    const auto supported_dims = std::make_tuple(
        std::make_tuple(std::integral_constant<int, 256>{}), // C
        std::make_tuple(std::integral_constant<int, 64>{}),  // H
        std::make_tuple(std::integral_constant<int, 64>{}),  // W
        std::make_tuple(std::integral_constant<int, 512>{}), // H_t
        std::make_tuple(std::integral_constant<int, 512>{})  // W_t
    );
    const auto runtime_dims = std::make_tuple(C, H, W, H_t, W_t);

    dispatch_kernel(static_launcher, runtime_dims, supported_dims);

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA error after forward kernel: ", hipGetErrorString(err));

    auto dice = (2.0 * total_intersection_sum + smooth) / (total_p_sum + total_t_sum + smooth);
    auto loss = (1.0 - dice).sum() / num_masks;

    return {loss, total_intersection_sum, total_p_sum, total_t_sum};
}

torch::Tensor mc_dice_loss_backward(
    const torch::Tensor& grad_out,
    const torch::Tensor& logits,
    const torch::Tensor& targets,
    const torch::Tensor& class_mapping,
    const torch::Tensor& total_intersection_sum,
    const torch::Tensor& total_p_sum,
    const torch::Tensor& total_t_sum,
    const float smooth
) {
    CHECK_INPUT(grad_out);
    CHECK_INPUT(logits);
    CHECK_INPUT(targets);
    CHECK_INPUT(total_intersection_sum);
    CHECK_INPUT(total_p_sum);
    CHECK_INPUT(total_t_sum);

    const int B = logits.size(0);
    const int C = logits.size(1);
    const int H = logits.size(2);
    const int W = logits.size(3);
    const int H_t = targets.size(1);
    const int W_t = targets.size(2);

    auto grad_logits = torch::empty_like(logits);
    if (logits.numel() == 0) return grad_logits;

    const float grad_out_scalar = grad_out.item<float>();

    dim3 grid(W, H, B);
    const size_t shared_mem_size = 2 * C * sizeof(int32_t);

    auto static_launcher = [&](auto... Dims) {
        dice_loss_backward_kernel<decltype(Dims)::value...><<<grid, THREADS_PER_BLOCK, shared_mem_size>>>(
            logits.data_ptr<float>(),
            targets.data_ptr<uint8_t>(),
            class_mapping.data_ptr<int64_t>(),
            total_intersection_sum.data_ptr<float>(),
            total_p_sum.data_ptr<float>(),
            total_t_sum.data_ptr<float>(),
            grad_out_scalar,
            grad_logits.data_ptr<float>(),
            B,
            smooth
        );
    };

    const auto supported_dims = std::make_tuple(
        std::make_tuple(std::integral_constant<int, 256>{}), // C
        std::make_tuple(std::integral_constant<int, 64>{}),  // H
        std::make_tuple(std::integral_constant<int, 64>{}),  // W
        std::make_tuple(std::integral_constant<int, 512>{}), // H_t
        std::make_tuple(std::integral_constant<int, 512>{})  // W_t
    );
    const auto runtime_dims = std::make_tuple(C, H, W, H_t, W_t);

    dispatch_kernel(static_launcher, runtime_dims, supported_dims);

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA error after backward kernel: ", hipGetErrorString(err));

    return grad_logits;
}