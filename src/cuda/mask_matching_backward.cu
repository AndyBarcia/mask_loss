#include "hip/hip_runtime.h"
#include <ATen/native/cuda/KernelUtils.cuh>
#include <c10/util/Half.h>
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <algorithm>
#include <cmath>
#include <tuple>
#include <type_traits>

#include "utils.h"
#include "utils.cuh"

namespace {

// CUDA kernels implementing the backward pass for the mask-matching loss.
// The kernels reuse the forward Hungarian assignments and accumulate
// sigmoid + dice gradients directly on device buffers.

// Threads used for computing gradients.
constexpr int GRAD_THREADS = 256;

__global__ void mask_matching_backward_kernel(
    const float* __restrict__ logits,
    const int64_t* __restrict__ matches,
    const uint8_t* __restrict__ counts,
    const float* __restrict__ grad_mask_mean,
    const float* __restrict__ grad_dice_mean,
    float* __restrict__ grad_logits,
    const int64_t L,
    const int64_t B,
    const int64_t C,
    const int64_t H,
    const int64_t W,
    const int64_t GT_out,
    const int64_t GT_total,
    const int64_t background_index,
    const float smooth,
    const float sigmoid_factor,
    const float dice_scale,
    const float area_scale,
    const float inv_denom
) {
    const int64_t g = blockIdx.x;
    const int64_t b = blockIdx.y;
    const int64_t l = blockIdx.z;

    if (l >= L || b >= B || g >= GT_out) {
        return;
    }

    const int64_t match_index = ((l * B) + b) * GT_out + g;
    const int64_t pred = matches[match_index];
    if (pred < 0 || pred >= C) {
        return;
    }

    int64_t actual_gt = g;
    if (background_index >= 0 && background_index < GT_total && g >= background_index) {
        actual_gt += 1;
    }
    if (actual_gt < 0 || actual_gt >= GT_total) {
        return;
    }

    // Each layer shares the same coefficients across ground-truth indices.
    const float mask_coeff = grad_mask_mean[l] * inv_denom;
    const float dice_coeff = grad_dice_mean[l] * inv_denom;
    if (mask_coeff == 0.0f && dice_coeff == 0.0f) {
        return;
    }

    const int64_t HW = H * W;
    const int64_t logits_base = (((l * B) + b) * C + pred) * H * W;
    const int64_t counts_base = (((b * GT_total) + actual_gt) * H) * W;

    __shared__ float sh_mask_sum;
    __shared__ float sh_target_sum;
    __shared__ float sh_inter_sum;
    if (threadIdx.x == 0) {
        sh_mask_sum = 0.0f;
        sh_target_sum = 0.0f;
        sh_inter_sum = 0.0f;
    }
    __syncthreads();

    float local_mask_sum = 0.0f;
    float local_target_sum = 0.0f;
    float local_inter_sum = 0.0f;

    for (int64_t idx = threadIdx.x; idx < HW; idx += blockDim.x) {
        const int64_t h = idx / W;
        const int64_t w = idx % W;

        const int64_t logits_offset = logits_base + h * W + w;
        const float logit = logits[logits_offset];
        const float prob = 1.0f / (1.0f + expf(-logit));
        const float prob_scaled = prob * area_scale;

        const int64_t counts_offset = counts_base + h * W + w;
        const float target = static_cast<float>(counts[counts_offset]);

        local_mask_sum += prob_scaled;
        local_target_sum += target;
        local_inter_sum += prob * target;
    }

    // Aggregate the per-thread partial sums for the dice statistics.
    atomicAdd(&sh_mask_sum, local_mask_sum);
    atomicAdd(&sh_target_sum, local_target_sum);
    atomicAdd(&sh_inter_sum, local_inter_sum);
    __syncthreads();

    const float mask_sum = sh_mask_sum;
    const float target_sum = sh_target_sum;
    const float inter_sum = sh_inter_sum;

    const float denom = mask_sum + target_sum + smooth;
    const float numerator = inter_sum * 2.0f + smooth;
    float denom_sq = denom * denom;
    if (denom_sq < 1e-20f) {
        denom_sq = 1e-20f;
    }
    const float inv_denom_sq = 1.0f / denom_sq;
    const float two_denom = denom * 2.0f;

    for (int64_t idx = threadIdx.x; idx < HW; idx += blockDim.x) {
        const int64_t h = idx / W;
        const int64_t w = idx % W;

        const int64_t logits_offset = logits_base + h * W + w;
        const float logit = logits[logits_offset];
        const float prob = 1.0f / (1.0f + expf(-logit));
        const float prob_scaled = prob * area_scale;
        const float prob_prime = prob * (1.0f - prob);

        const int64_t counts_offset = counts_base + h * W + w;
        const float target = static_cast<float>(counts[counts_offset]);

        // Sigmoid CE gradient and dice gradient for a single pixel.
        const float grad_sigmoid = (prob_scaled - target) * sigmoid_factor;
        const float d_inter = prob_prime * target;
        const float d_denom = prob_prime * area_scale;
        const float grad_dice = (numerator * d_denom - two_denom * d_inter) * inv_denom_sq * dice_scale;

        const float grad = mask_coeff * grad_sigmoid + dice_coeff * grad_dice;
        grad_logits[logits_offset] = grad;
    }
}

} // namespace

torch::Tensor mask_matching_backward(
    const torch::Tensor& grad_layer_mask_mean,
    const torch::Tensor& grad_layer_dice_mean,
    const torch::Tensor& logits,
    const torch::Tensor& targets,
    const torch::Tensor& matches,
    const float smooth,
    const float sigmoid_scale,
    const float dice_scale,
    const int64_t background_index,
    const int64_t num_masks,
    const int64_t matched_count
) {
    // Backward pipeline:
    //   1. Materialize downsampled ground-truth masks once for reuse.
    //   2. Launch one CUDA block per (layer, batch, gt) assignment to compute
    //      sigmoid and dice gradients directly into the logits tensor.
    CHECK_INPUT(grad_layer_mask_mean);
    CHECK_INPUT(grad_layer_dice_mean);
    CHECK_INPUT(logits);
    CHECK_INPUT(targets);
    CHECK_INPUT(matches);

    const auto device = logits.device();
    TORCH_CHECK(targets.device() == device, "targets must be on the same device as logits");
    TORCH_CHECK(matches.device() == device, "matches must be on the same device as logits");

    TORCH_CHECK(logits.scalar_type() == torch::kFloat32, "logits must be float32");
    TORCH_CHECK(grad_layer_mask_mean.scalar_type() == torch::kFloat32, "grad_layer_mask_mean must be float32");
    TORCH_CHECK(grad_layer_dice_mean.scalar_type() == torch::kFloat32, "grad_layer_dice_mean must be float32");

    TORCH_CHECK(targets.scalar_type() == torch::kLong, "logits must be long");
    TORCH_CHECK(logits.is_contiguous(), "logits must be contiguous");
    TORCH_CHECK(matches.is_contiguous(), "matches must be contiguous");
    TORCH_CHECK(grad_layer_mask_mean.is_contiguous(), "grad_layer_mask_mean must be contiguous");
    TORCH_CHECK(grad_layer_dice_mean.is_contiguous(), "grad_layer_dice_mean must be contiguous");

    const int64_t L = logits.size(0);
    const int64_t B = logits.size(1);
    const int64_t C = logits.size(2);
    const int64_t H = logits.size(3);
    const int64_t W = logits.size(4);

    const int64_t H_t = targets.size(1);
    const int64_t W_t = targets.size(2);

    TORCH_CHECK(H_t % H == 0 && W_t % W == 0, "Target resolution must be an integer multiple of logits resolution.");
    const int64_t scale = H_t / H;
    TORCH_CHECK(scale > 0, "Invalid spatial scale");
    const float area_scale = static_cast<float>(scale * scale);

    const int64_t GT_out = matches.size(2);
    TORCH_CHECK(GT_out > 0 || targets.numel() == 0, "matches must have a non-zero last dimension");

    auto grad_logits = torch::zeros_like(logits);

    // The forward pass counts valid assignments while aggregating losses, so we
    // can reuse the same value here without scanning the matches tensor again.
    if (matched_count <= 0) {
        return grad_logits;
    }

    // Derive the normalization factor used for both dice and sigmoid terms.
    int64_t denom_masks = num_masks > 0 ? num_masks : matched_count;
    if (denom_masks <= 0) {
        denom_masks = 1;
    }
    const float inv_denom = 1.0f / static_cast<float>(denom_masks);

    int64_t GT_total = 0;
    if (targets.numel() > 0) {
        GT_total = targets.max().item<int64_t>() + 1;
    }
    if (GT_total == 0) {
        return grad_logits;
    }

    // Pre-compute the per-(batch,gt) downsampled label counts so each CUDA block
    // can reuse them without touching the targets tensor again.
    auto counts = torch::zeros({B, GT_total, H, W}, logits.options().dtype(torch::kUInt8));

    {
        dim3 block(16, 16);
        dim3 grid((W + block.x - 1) / block.x, (H + block.y - 1) / block.y, B);

        auto static_launcher = [&](auto H_val, auto W_val, auto H_t_val, auto W_t_val) {
            count_labels_per_block_kernel<
                decltype(H_val)::value,
                decltype(W_val)::value,
                decltype(H_t_val)::value,
                decltype(W_t_val)::value><<<grid, block>>>(
                    targets.data_ptr<int64_t>(),
                    counts.data_ptr<uint8_t>(),
                    static_cast<int>(B),
                    static_cast<int>(GT_total)
                );
        };

        const auto runtime_dims = std::make_tuple(static_cast<int>(H), static_cast<int>(W), static_cast<int>(H_t), static_cast<int>(W_t));
        const auto supported_dims = std::make_tuple(
            std::make_tuple(std::integral_constant<int, 256>{}),
            std::make_tuple(std::integral_constant<int, 256>{}),
            std::make_tuple(std::integral_constant<int, 1024>{}),
            std::make_tuple(std::integral_constant<int, 1024>{})
        );
        dispatch_kernel(static_launcher, runtime_dims, supported_dims);
        CHECK_CUDA_ERROR(hipGetLastError());
    }

    const float norm = 1.0f / static_cast<float>(H_t * W_t);
    const float sigmoid_factor = sigmoid_scale * norm;

    if (L == 0 || B == 0 || GT_out == 0) {
        return grad_logits;
    }

    dim3 grad_grid(static_cast<unsigned int>(GT_out), static_cast<unsigned int>(B), static_cast<unsigned int>(L));
    mask_matching_backward_kernel<<<grad_grid, GRAD_THREADS>>>(
        logits.data_ptr<float>(),
        matches.data_ptr<int64_t>(),
        counts.data_ptr<uint8_t>(),
        grad_layer_mask_mean.data_ptr<float>(),
        grad_layer_dice_mean.data_ptr<float>(),
        grad_logits.data_ptr<float>(),
        L,
        B,
        C,
        H,
        W,
        GT_out,
        GT_total,
        background_index,
        smooth,
        sigmoid_factor,
        dice_scale,
        area_scale,
        inv_denom
    );
    CHECK_CUDA_ERROR(hipGetLastError());

    return grad_logits;
}

