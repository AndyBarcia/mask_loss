#include "hip/hip_runtime.h"
#include <ATen/native/cuda/KernelUtils.cuh>
#include <c10/util/Half.h>
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <cmath>

#include "utils.h"
#include "utils.cuh"

// Kernel: for each (l,b,q) reduce across C once to get sum_neg,
// then emit (optionally compacted) GT_out values using the one-hot identity:
// BCE(one-hot(y)) = sum_c neg(z_c) - z_y, where
// neg(z) = max(z,0) + log1p(exp(-|z|))
template <int C>
__global__ void __launch_bounds__(REDUCTION_THREADS_PER_BLOCK)
reduce_pairwise_label_kernel(
    const float* __restrict__ logits,      // (L, B, Q, C)
    const int64_t* __restrict__ targets,   // (B, GT_total)
    float* __restrict__ out,               // (L, B, Q, GT_out)
    const int32_t background_index,        // fixed column to drop; set to GT_total if none
    const int32_t GT_total,                // number of GT slots (columns in targets)
    const int32_t GT_out,                  // GT_total - (background dropped ? 1 : 0)
    const int32_t B,
    const int32_t Q,
    const int32_t L,
    const float scale,
    const float gamma,
    const float alpha
) {
    constexpr int NUM_WARPS = REDUCTION_THREADS_PER_BLOCK / 32;
    __shared__ float s_warp[NUM_WARPS];

    const int l   = blockIdx.x;  // layer
    const int b   = blockIdx.y;  // batch
    const int qid = blockIdx.z;  // query
    const int tid = threadIdx.x;

    // Reduce across C to get sum_neg(l,b,q)
    float thread_sum = 0.f;
    const float alpha_pos = (alpha >= 0.0f) ? alpha : 1.0f;
    const float alpha_neg = (alpha >= 0.0f) ? (1.0f - alpha) : 1.0f;
    const bool use_gamma = (gamma > 0.0f);

    // Stride across C by blockDim.x; template C is compile-time for efficient looping
    for (int c = tid; c < C; c += REDUCTION_THREADS_PER_BLOCK) {
        const float z = logits[(((l * B + b) * Q + qid) * C) + c];
        const float maxL  = z > 0.f ? z : 0.f;
        const float absL  = fabsf(z);
        const float logex = log1pf(__expf(-absL));
        const float ce_neg = logex + maxL;
        const float sigma  = 1.0f / (1.0f + __expf(-z));
        const float mod_neg = use_gamma ? powf(sigma, gamma) : 1.0f;
        thread_sum += alpha_neg * mod_neg * ce_neg;
    }

    // Warp reduce to a single value per block
    float base_sum = thread_sum;
    #pragma unroll
    for (int off = 16; off > 0; off >>= 1) {
        base_sum += __shfl_down_sync(0xffffffff, base_sum, off);
    }
    if ((tid & 31) == 0) s_warp[tid >> 5] = base_sum;
    __syncthreads();
    for (int s = NUM_WARPS >> 1; s > 0; s >>= 1) {
        if (tid < s) s_warp[tid] += s_warp[tid + s];
        __syncthreads();
    }
    base_sum = (tid == 0) ? s_warp[0] : 0.f;
    // Broadcast to all threads in warp 0 (for symmetry; only tid==0 will write outputs)
    base_sum = __shfl_sync(0xffffffff, base_sum, 0);

    // For each output GT slot, write loss or +inf for padding ---
    // We only need one writer (tid==0) since (l,b,q,*) are independent
    if (tid == 0) {
        const float invC = (C > 0) ? (1.f / static_cast<float>(C)) : 0.f;

        for (int out_gt_idx = 0; out_gt_idx < GT_out; ++out_gt_idx) {
            const int gt_actual = MAP_OUT_TO_ACTUAL(out_gt_idx, background_index);
            const int64_t y64   = targets[b * GT_total + gt_actual];

            // Padding / invalid label -> +inf (do not apply scale, mirroring other kernel)
            if (y64 < 0 || y64 >= static_cast<int64_t>(C)) {
                out[(((l * B + b) * Q + qid) * GT_out) + out_gt_idx] = INFINITY;
                continue;
            }

            const int y = static_cast<int>(y64);
            const float z_val = logits[(((l * B + b) * Q + qid) * C) + y];
            const float absL  = fabsf(z_val);
            const float maxL  = z_val > 0.f ? z_val : 0.f;
            const float maxNeg = (-z_val) > 0.f ? -z_val : 0.f;
            const float logex = log1pf(__expf(-absL));
            const float ce_neg = logex + maxL;
            const float ce_pos = logex + maxNeg;
            const float sigma  = 1.0f / (1.0f + __expf(-z_val));
            const float one_minus = 1.0f - sigma;
            const float mod_neg = use_gamma ? powf(sigma, gamma) : 1.0f;
            const float mod_pos = use_gamma ? powf(one_minus, gamma) : 1.0f;
            const float delta = alpha_pos * mod_pos * ce_pos - alpha_neg * mod_neg * ce_neg;
            const float v = (base_sum + delta) * invC;
            out[(((l * B + b) * Q + qid) * GT_out) + out_gt_idx] = v * scale;
        }
    }
}

torch::Tensor pairwise_label_loss_forward(
    const torch::Tensor& logits,   // (L,B,Q,C), float
    const torch::Tensor& targets,  // (B,GT), int64 with -1 padding
    int64_t background_index = -1, // drop column targets[:, background_index]
    const float scale = 1.0f,
    const float gamma = 0.0f,
    const float alpha = -1.0f
) {
    CHECK_INPUT(logits);
    CHECK_INPUT(targets);

    TORCH_CHECK(logits.dim() == 4, "pairwise_label_loss_forward: logits must be (L,B,Q,C)");
    TORCH_CHECK(targets.dim() == 2, "pairwise_label_loss_forward: targets must be (B,GT)");
    TORCH_CHECK(gamma >= 0.0f, "pairwise_label_loss_forward: focal_gamma must be non-negative");
    TORCH_CHECK(alpha < 0.0f || (alpha >= 0.0f && alpha <= 1.0f),
                "pairwise_label_loss_forward: focal_alpha must be in [0,1] or negative to disable");

    const int L  = static_cast<int>(logits.size(0));
    const int B  = static_cast<int>(logits.size(1));
    const int Q  = static_cast<int>(logits.size(2));
    const int C  = static_cast<int>(logits.size(3));
    const int GT_total = static_cast<int>(targets.size(1));

    TORCH_CHECK(B == targets.size(0), "pairwise_label_loss_forward: batch size mismatch between logits and targets");

    // Determine whether to drop a fixed GT column across the batch
    const bool drop_bg_col = (background_index >= 0 && background_index < GT_total);
    if (background_index < 0) {
        // Set to an invalid index so device-side MAP_OUT_TO_ACTUAL is a no-op
        background_index = GT_total;
    }
    const int GT_out = GT_total - (drop_bg_col ? 1 : 0);

    // Edge case: nothing to compute
    if (GT_out == 0) {
        return torch::zeros({L, B, Q, 0}, logits.options().dtype(torch::kFloat32));
    }

    // Ensure dtype/layout
    auto logits_f = logits.contiguous().to(torch::kFloat32);
    auto targets_i64 = targets.contiguous(); // keep int64

    // Allocate output
    auto out = torch::empty({L, B, Q, GT_out}, logits.options().dtype(torch::kFloat32));

    // Launch kernel: one block per (l,b,q), reduce across C
    dim3 grid(L, B, Q);

    auto static_launcher = [&](auto C_val) {
        reduce_pairwise_label_kernel<decltype(C_val)::value>
            <<<grid, REDUCTION_THREADS_PER_BLOCK>>>(
                logits_f.data_ptr<float>(),
                targets_i64.data_ptr<int64_t>(),
                out.data_ptr<float>(),
                static_cast<int32_t>(background_index),
                static_cast<int32_t>(GT_total),
                static_cast<int32_t>(GT_out),
                static_cast<int32_t>(B),
                static_cast<int32_t>(Q),
                static_cast<int32_t>(L),
                scale,
                gamma,
                alpha
            );
    };

    // Template-dispatch over C for performance (matches your style)
    const auto supported_dims = std::make_tuple(
        std::make_tuple(std::integral_constant<int, 128>{}) // C
    );
    const auto runtime_dims = std::make_tuple(C);
    dispatch_kernel(static_launcher, runtime_dims, supported_dims);

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess,
                "CUDA error in pairwise_label_loss_forward kernel: ",
                hipGetErrorString(err));

    return out;
}
